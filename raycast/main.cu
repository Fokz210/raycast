#include "hip/hip_runtime.h"
////////////////////////////////
//-----------CUDA-------------//
////////////////////////////////
#include <hip/hip_runtime.h>             
#include <>
#include <hiprand.h>

////////////////////////////////
//-----------SFML-------------//
////////////////////////////////
#include <SFML/Graphics.hpp>

////////////////////////////////
//-----------STD--------------//
////////////////////////////////
#include <iostream>
#include <numeric>

////////////////////////////////
//-----------LOCAL------------//
////////////////////////////////
#include "sfml_context.h"
#include "sphere.h"
#include "camera.h"

//==============================================================

__device__ vector3f ray_cast (ray const & r, sphere const & sph) noexcept;
__global__ void render (sfml_context::color * const colorbuffer, int const width, int const height, sphere * sphs, int const sphc, camera const cam);

int const threads = 32;

int main ()
{
	sfml_context window (1920, 1080);

	window.clear ();

	sfml_context::color * device_mem;

	hipMalloc (&device_mem, window.width () * window.height () * sizeof (sfml_context::color));

	float phi = 0.f;
	float theta = 0.4f;
	
	sphere sphs[2] = { sphere (vector3f (0.f, 0.f, 0.3f), 0.1f, vector3f (1.f, 1.f, 1.f)),
					   sphere (vector3f (0.f, 0.f, -1000.f), 1000.f, vector3f (0.3f, 0.7f, 0.3f)) };

	sphere * sphs_vptr = nullptr;
	hipMalloc (&sphs_vptr, sizeof (sphere) * 2);
	hipMemcpy (sphs_vptr, sphs, sizeof (sphere) * 2, hipMemcpyHostToDevice);

	while (window.is_open())
	{
		float const s = 2e-3;

		phi = static_cast <float> (sf::Mouse::getPosition ().x) * s;
		theta = static_cast <float> (sf::Mouse::getPosition ().y) * s;

		float const dist = 2.f;

		camera const cam
		(
			1.f,
			static_cast <float> (window.width ()) / window.height (),
			vector3f (std::cos (phi) * std::cos (theta), std::sin (phi) * std::cos (theta), std::sin (theta)) * dist,
			vector3f (0.f, 0.f, 1.f),
			vector3f (0.f, 0.f, 0.3f)
		);

		render <<< window.width() * window.height() / threads, threads >>> (device_mem, window.width (), window.height (), sphs_vptr, 2, cam);
		hipMemcpy (window.memory (), device_mem, window.width () * window.height () * sizeof (sfml_context::color), hipMemcpyDeviceToHost);
		window.update ();
	}
	
	hipFree (device_mem);
	return 0;
}

__device__ vector3f ray_cast (ray const & r, sphere * sphs, int sphere_count) noexcept
{
	float dist = 200000;

	sphere::intersection in{ false, SKY_COLOR, vector3f (), vector3f () };

	for (int i = 0; i < sphere_count; i++)
	{
		sphere::intersection new_intersection = sphs[i].ray_intersect (r, dist);

		if (new_intersection.happened && (in.happened && dot (in.pos - r.pos, in.pos - r.pos) > dot (new_intersection.pos - r.pos, new_intersection.pos - r.pos) || !in.happened))
		{
			in = new_intersection;
		}
	}

	return in.color;
}

__global__ void render (sfml_context::color * const colorbuffer, int const width, int const height, sphere * sphs, int const sphc, camera const cam)
{
	int const i = blockIdx.x * blockDim.x + threadIdx.x;
	float2 const p = { i % width, i / width };
	int const j = p.x + width * (height - p.y - 1);
	float2 const f = { (2.f * p.x) / width - 1.f, (2.f * p.y) / height - 1.f };

	vector3f color = ray_cast (cam.cast (f.x, f.y), sphs, sphc);

	colorbuffer[j] = RGBA8
	{
		static_cast <sf::Uint8> (255 * color.x),
		static_cast <sf::Uint8> (255 * color.y),
		static_cast <sf::Uint8> (255 * color.z),
		255
	};
}

__device__ vector3f hemi_vector3f (vector3f const & norm, float const rand1, float const rand2)
{
	vector3f const x0 = cross (norm, vector3f (0.f, 0.f, 1.f));
	vector3f x;
	if (dot (x0, x0) < 1e-4)
		x = normalize (cross (norm, vector3f (0.f, 1.f, 0.f)));
	else
		x = normalize (x0);
	vector3f const y = cross (norm, x);

	float const cos_theta = sqrtf (rand1);
	float const sin_theta = sqrtf (1.f - rand1);
	float const phi = rand2 * 3.14159265f * 2.f;
	return x * cos_theta * cos (phi) + y * cos_theta * sin (phi) + norm * sin_theta;
}
